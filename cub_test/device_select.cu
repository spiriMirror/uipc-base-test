#include <hip/hip_runtime.h>
#include <cub/device/device_select.cuh>
#include <thrust/device_vector.h>
#include <iostream>

void test_device_select(size_t count) {

	std::cout << "Testing DeviceSelect::Flag: Count=" << count << std::endl;

	thrust::device_vector<int> d_in(count);
	thrust::device_vector<int> d_out(count);
	thrust::device_vector<bool> d_flags(count);
	thrust::device_vector<int> d_out_count(1);
	size_t temp_storage_bytes = 0;
	thrust::device_vector<char> d_temp_storage;
	hipcub::DeviceSelect::Flagged(thrust::raw_pointer_cast(d_temp_storage.data()),
		temp_storage_bytes,
		thrust::raw_pointer_cast(d_in.data()),
		thrust::raw_pointer_cast(d_out.data()),
		thrust::raw_pointer_cast(d_out_count.data()),
		thrust::raw_pointer_cast(d_flags.data()),
		count);
	d_temp_storage.resize(temp_storage_bytes);
	auto F = hipcub::DeviceSelect::Flagged(thrust::raw_pointer_cast(d_temp_storage.data()),
		temp_storage_bytes,
		thrust::raw_pointer_cast(d_in.data()),
		thrust::raw_pointer_cast(d_out.data()),
		thrust::raw_pointer_cast(d_out_count.data()),
		thrust::raw_pointer_cast(d_flags.data()),
		count);

	if (F != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(F) << std::endl;
	}
	else {
		std::cout << "Success: " << count << " elements processed." << std::endl;
	}
}

int main()
{
	for (size_t count = 1024; count <= 1024 * 1024; count *= 2) {
		test_device_select(count);
	}
}